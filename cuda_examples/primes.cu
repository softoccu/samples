#include <iostream>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256  // Number of threads per block

// CUDA kernel to mark multiples of a given prime
__global__ void sieve_kernel(bool* primes, int start, int end, int prime, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Mark multiples of prime within the range [start, end]
    if (idx * prime + start <= end && (idx * prime + start) <= n) {
        primes[idx * prime + start] = false;
    }
}

// Function to find primes using CUDA
void find_primes(int n) {
    // Array to hold the prime status of numbers
    // you may use bitset to save memory, here to show how to use CUDA, just keep the code simple
    bool *primes = new bool[n + 1];  
    for (int i = 0; i <= n; ++i) {
        primes[i] = true;
    }
    primes[0] = primes[1] = false;  // 0 and 1 are not prime

    // Allocate memory on GPU
    bool* d_primes;
    hipMalloc(&d_primes, (n + 1) * sizeof(bool));
    hipMemcpy(d_primes, primes, (n + 1) * sizeof(bool), hipMemcpyHostToDevice);

    int sqrt_n = sqrt(n);

    for (int prime = 2; prime <= sqrt_n; ++prime) {
        if (primes[prime]) {
            int start = prime * prime;
            int end = n;

            // Launch kernel to mark multiples of prime as false (non-prime)
            int num_threads = (end - start) / prime + 1;
            int num_blocks = (num_threads + BLOCK_SIZE - 1) / BLOCK_SIZE;

            sieve_kernel<<<num_blocks, BLOCK_SIZE>>>(d_primes, start, end, prime, n);
        }
    }

    // Copy result back to host
    hipMemcpy(primes, d_primes, (n + 1) * sizeof(bool), hipMemcpyDeviceToHost);

    // Print primes
    for (int i = 2; i <= n; ++i) {
        if (primes[i]) {
            std::cout << i << " ";
        }
    }
    std::cout << std::endl;

    // Free allocated memory
    delete[] primes;
    hipFree(d_primes);
}

int main() {
    int N = 1000000;  // Limit for prime search, you can adjust this number
    find_primes(N);    // Call the function to find primes
    return 0;
}

