#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

// Macro for error checking CUDA calls
#define CHECK_CUDA(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// Macro for error checking cuFFT calls
#define CHECK_CUFFT(call) \
    do { \
        hipfftResult err = call; \
        if (err != HIPFFT_SUCCESS) { \
            std::cerr << "CUFFT error: " << err << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// FFT example
int main() {
    const int NX = 256;  // Size of input data
    const int BATCH = 1; // Batch size

    // Allocate host memory
    hipfftComplex *h_data = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX * BATCH);

    // Initialize input data
    for (int i = 0; i < NX; ++i) {
        h_data[i].x = static_cast<float>(i);
        h_data[i].y = 0.0f;
    }

    // Allocate device memory
    hipfftComplex *d_data;
    CHECK_CUDA(hipMalloc((void **)&d_data, sizeof(hipfftComplex) * NX * BATCH));

    // Copy data from host to device
    CHECK_CUDA(hipMemcpy(d_data, h_data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice));

    // Create cuFFT plan
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH));

    // Execute FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));

    // Copy result from device back to host
    CHECK_CUDA(hipMemcpy(h_data, d_data, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyDeviceToHost));

    // Output result
    for (int i = 0; i < NX; ++i) {
        std::cout << "Result[" << i << "] = (" << h_data[i].x << ", " << h_data[i].y << ")" << std::endl;
    }

    // Clean up resources
    CHECK_CUFFT(hipfftDestroy(plan));
    CHECK_CUDA(hipFree(d_data));
    free(h_data);

    return 0;
}